﻿
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <fstream>

#define NOTHING 0
#define NEW 1
#define BURN1 2
#define BURN2 3
#define BURNT 4

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);
hipError_t simStep(int* c);

const int chamberWidth = 200;
const int chamberHeight = 200;
const int chamberArrLen = chamberHeight * chamberWidth;

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
    printf("%d", i);
}

__global__ void simStepKernel(int* c) {
    int i = threadIdx.x;
}

__global__ void simKernel(int* chamber) {
    int x = threadIdx.x;
    int y = threadIdx.y;
    int cellnumber = (y * chamberWidth) + x;
    switch (chamber[cellnumber]) {
    case NOTHING:
        return;
        break;
    case NEW:
        if (chamber[cellnumber + chamberWidth] == BURN2 || chamber[cellnumber + 1] == BURN2) {
            chamber[cellnumber] = BURN1;
        }
        return;
        break;
    case BURN1:
        chamber[cellnumber] == BURN2;
        return;
        break;
    default:
        break;
    }
}

int main()
{
    std::ofstream img("picture.ppm");
    img << "P3" << std::endl;
    img << "200 200" << std::endl;
    img << "255" << std::endl;

    for (int y = 0; y < 200; y++) {
        for (int x = 0; x < 200; x++){
            img << "0 120 30" << std::endl;
        }
    }

    system("cd C:\\Users\\morit\\source\\repos\\BrennkammerSim\\");
    system("magick convert picture.ppm picture.png");


    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
